#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>

__global__ void rgba_to_grey(unsigned char *d_grey, uchar4 *d_rgba,
                             const std::size_t rows, const std::size_t cols) {
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int idy = threadIdx.y + blockDim.y * blockIdx.y;

    if (idx >= rows || idy >= cols) return;

    uchar4 p = d_rgba[idx * cols + idy];
    d_grey[idx * cols + idy] =
        (unsigned char)(0.299f * p.x + 0.587f * p.y + 0.114f * p.z);
}

using namespace cv;

// parameter parsing
// pre kernel processing
// kernel processing
// post kernel processing
int main(int argc, char *argv[]) {
    clock_t start_t, end_t;
    double total_t;

    std::string inimg = std::string(argv[1]);
    std::string outimg = std::string(argv[2]);
    cv::Mat img = cv::imread(inimg);

    cv::Mat img_rgba, img_grey;
    cv::cvtColor(img, img_rgba, cv::COLOR_RGBA2BGRA);
    img_grey.create(img_rgba.rows, img_rgba.cols, CV_8UC1);

    uchar4 *h_rgba = (uchar4 *)img_rgba.ptr<unsigned char>(0);
    unsigned char *h_grey = img_grey.ptr<unsigned char>(0);

    uchar4 *d_rgba;
    unsigned char *d_grey;

    unsigned int np = img_rgba.rows * img_rgba.cols;
    hipMalloc((void **)&d_rgba, sizeof(uchar4) * np);
    hipMalloc((void **)&d_grey, sizeof(unsigned char) * np);
    hipMemset((void **)d_grey, 0, sizeof(unsigned char) * np);
    hipMemcpy((void **)d_rgba, (void **)h_rgba, sizeof(uchar4) * np,
               hipMemcpyHostToDevice);

    std::size_t rows = img_rgba.rows;
    std::size_t cols = img_rgba.cols;

    const std::size_t BLOCK_WIDTH = 32;

    const dim3 block_size(BLOCK_WIDTH, BLOCK_WIDTH, 1);
    unsigned int grid_x = (unsigned int)(rows / BLOCK_WIDTH + 1);
    unsigned int grid_y = (unsigned int)(cols / BLOCK_WIDTH + 1);
    const dim3 grid_size(grid_x, grid_y, 1);

    start_t = clock();
    rgba_to_grey<<<grid_size, block_size>>>(d_grey, d_rgba, rows, cols);
    end_t = clock();

    hipMemcpy(h_grey, d_grey, sizeof(unsigned char) * np,
               hipMemcpyDeviceToHost);

    cv::imwrite(outimg.c_str(), img_grey);

    // cv::imshow("rgb", img_rgba);
    // cv::imshow("grey", img_grey);
    // cv::waitKey(0);

    total_t = (double)(end_t - start_t) / CLOCKS_PER_SEC;
    printf("Total time taken by GPU+CPU: %f\n", total_t);

    hipFree(d_rgba);
    hipFree(d_grey);

    return 0;
}
